#include "macros.cuh"

inline void gpuAssert(hipError_t code, const char *file, int line) {
    if (code != hipSuccess) {
        fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file,
                line);
    }
}