
#include <hip/hip_runtime.h>
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort) {
    if (code != hipSuccess) {
        fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file,
                line);
        if (abort) {
            exit(code);
        }
    }
}