#include "hip/hip_runtime.h"
#include <algorithm>

#include "search.cuh"
#include "macros.cuh"
#include "evaluate.cuh"
#include "moves.cuh"

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }

inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

int * h_level_sizes;
int * d_level_sizes;
int * h_subtree_sizes;
int * d_subtree_sizes;
int * last;

__device__ void gather_results(int* results_to, int* results_from, bool maximize, int * last) {
    int result;
    if (maximize) { // maximizing
        result = -INF;
        for (int i = 0; i < BOARDS_GENERATED; i++) {
            if (results_from[i] != INF && results_from[i] != -INF && results_from[i] > result) {
                result = results_from[i];
                *last = i;
            }
        }
    } 
    else { // minimizing
        result = INF;
        for (int i = 0; i < BOARDS_GENERATED; i++) {
           if (results_from[i] != INF && results_from[i] != -INF && results_from[i] < result) {
                result = results_from[i];
                *last = i;
            }
        }
    }
    DBG(printf("Zebralem wyniki i mam %d\n", result));
    *results_to = result;
}  

__device__ __host__ void _init_sizes_tables(int* level_sizes, int * subtree_sizes) {
    level_sizes[0] = 1;
    subtree_sizes[0] = 1;
    for (int i = 1;  i <= MAX_DEPTH; i++) {
        level_sizes[i] = level_sizes[i - 1] * BOARDS_GENERATED;
        subtree_sizes[i] = level_sizes[i - 1] * BOARDS_GENERATED + subtree_sizes[i - 1];
    }
}

__global__ void init_sizes_tables(int* level_sizes, int * subtree_sizes) {
    _init_sizes_tables(level_sizes, subtree_sizes);   
}

void init() {
    h_level_sizes = new int[MAX_DEPTH + 1];
    h_subtree_sizes = new int[MAX_DEPTH + 2];
    CHECK_ALLOC(hipMalloc(&d_level_sizes, sizeof(int) * MAX_DEPTH + 1));
    CHECK_ALLOC(hipMalloc(&d_subtree_sizes, sizeof(int) * MAX_DEPTH + 2));
    CHECK_ALLOC(hipMalloc(&last, sizeof(int)));

    init_sizes_tables<<<1, 1>>>(d_level_sizes, d_subtree_sizes);
    _init_sizes_tables(h_level_sizes, h_subtree_sizes);
    gpuErrchk(hipDeviceSynchronize());
    gpuErrchk(hipPeekAtLastError());
}

void terminate() {
    free(h_level_sizes);
    free(h_subtree_sizes);
    hipFree(d_level_sizes);
    hipFree(d_subtree_sizes);
    hipFree(last);
}

__global__ void init_searching(pos64 * white_pawns_boards,
                    pos64 * white_bishops_boards,
                    pos64 * white_knights_boards,
                    pos64 * white_rooks_boards,
                    pos64 * white_queens_boards,
                    pos64 * white_kings_boards,
                    pos64 * black_pawns_boards,
                    pos64 * black_bishops_boards,
                    pos64 * black_knights_boards,
                    pos64 * black_rooks_boards,
                    pos64 * black_queens_boards,
                    pos64 * black_kings_boards,
                    const pos64 white_pawns,
                    const pos64 white_bishops,
                    const pos64 white_knights,
                    const pos64 white_rooks,
                    const pos64 white_queens,
                    const pos64 white_kings,
                    const pos64 black_pawns,
                    const pos64 black_bishops,
                    const pos64 black_knights,
                    const pos64 black_rooks,
                    const pos64 black_queens,
                    const pos64 black_kings) {
    white_pawns_boards[0]   = white_pawns;
    white_bishops_boards[0] = white_bishops;
    white_knights_boards[0] = white_knights;
    white_rooks_boards[0]   = white_rooks;
    white_queens_boards[0]  = white_queens;
    white_kings_boards[0]   = white_kings;
    black_pawns_boards[0]   = black_pawns;
    black_bishops_boards[0] = black_bishops;
    black_knights_boards[0] = black_knights;
    black_rooks_boards[0]   = black_rooks;
    black_queens_boards[0]  = black_queens;
    black_kings_boards[0]   = black_kings;
}

void end_searching(pos64 * white_pawns,
                    pos64 * white_bishops,
                    pos64 * white_knights,
                    pos64 * white_rooks,
                    pos64 * white_queens,
                    pos64 * white_kings,
                    pos64 * black_pawns,
                    pos64 * black_bishops,
                    pos64 * black_knights,
                    pos64 * black_rooks,
                    pos64 * black_queens,
                    pos64 * black_kings,
                    pos64 * white_pawns_boards,
                    pos64 * white_bishops_boards,
                    pos64 * white_knights_boards,
                    pos64 * white_rooks_boards,
                    pos64 * white_queens_boards,
                    pos64 * white_kings_boards,
                    pos64 * black_pawns_boards,
                    pos64 * black_bishops_boards,
                    pos64 * black_knights_boards,
                    pos64 * black_rooks_boards,
                    pos64 * black_queens_boards,
                    pos64 * black_kings_boards,
                    int * d_last) {
    int last;
    hipMemcpy(&last, d_last, sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(white_pawns, white_pawns_boards + last + 1, sizeof(pos64), hipMemcpyDeviceToHost);
    hipMemcpy(white_bishops, white_bishops_boards + last + 1, sizeof(pos64), hipMemcpyDeviceToHost);
    hipMemcpy(white_knights, white_knights_boards + last + 1, sizeof(pos64), hipMemcpyDeviceToHost);
    hipMemcpy(white_rooks, white_rooks_boards + last + 1, sizeof(pos64), hipMemcpyDeviceToHost);
    hipMemcpy(white_queens, white_queens_boards + last + 1, sizeof(pos64), hipMemcpyDeviceToHost);
    hipMemcpy(white_kings, white_kings_boards + last + 1, sizeof(pos64), hipMemcpyDeviceToHost);
    hipMemcpy(black_pawns, black_pawns_boards + last + 1, sizeof(pos64), hipMemcpyDeviceToHost);
    hipMemcpy(black_bishops, black_bishops_boards + last + 1, sizeof(pos64), hipMemcpyDeviceToHost);
    hipMemcpy(black_knights, black_knights_boards + last + 1, sizeof(pos64), hipMemcpyDeviceToHost);
    hipMemcpy(black_rooks, black_rooks_boards + last + 1, sizeof(pos64), hipMemcpyDeviceToHost);
    hipMemcpy(black_queens, black_queens_boards + last + 1, sizeof(pos64), hipMemcpyDeviceToHost);
    hipMemcpy(black_kings, black_kings_boards + last + 1, sizeof(pos64), hipMemcpyDeviceToHost);
}


__global__ void run_first_stage(pos64 * white_pawns_boards,
                pos64 * white_bishops_boards,
                pos64 * white_knights_boards,
                pos64 * white_rooks_boards,
                pos64 * white_queens_boards,
                pos64 * white_kings_boards,
                pos64 * black_pawns_boards,
                pos64 * black_bishops_boards,
                pos64 * black_knights_boards,
                pos64 * black_rooks_boards,
                pos64 * black_queens_boards,
                pos64 * black_kings_boards,
                int level,
                short current_player,
                int * level_sizes,
                int * subtree_sizes,
                int basic_offset = 0) {
    int index = blockIdx.x * 1024 + threadIdx.x;
    if (index >= level_sizes[level]) return;
    int index_offset = (level == 0 ? 0 : subtree_sizes[level - 1]) + basic_offset;
    int kids_offset = subtree_sizes[level] + index * BOARDS_GENERATED + basic_offset;
    current_player ^= (level & 1);

    DBG(if (index != 0) {return;})
    DBG(printf("Generuje ruchy gracza %d od pozycji %d i current_player %d\n", index + index_offset, kids_offset, current_player));

    generate_moves(&white_pawns_boards[index + index_offset],
                    &white_bishops_boards[index + index_offset],
                    &white_knights_boards[index + index_offset],
                    &white_rooks_boards[index + index_offset],
                    &white_queens_boards[index + index_offset],
                    &white_kings_boards[index + index_offset],
                    &black_pawns_boards[index + index_offset],
                    &black_bishops_boards[index + index_offset],
                    &black_knights_boards[index + index_offset],
                    &black_rooks_boards[index + index_offset],
                    &black_queens_boards[index + index_offset],
                    &black_kings_boards[index + index_offset],
                    &white_pawns_boards[kids_offset],
                    &white_bishops_boards[kids_offset],
                    &white_knights_boards[kids_offset],
                    &white_rooks_boards[kids_offset],
                    &white_queens_boards[kids_offset],
                    &white_kings_boards[kids_offset],
                    &black_pawns_boards[kids_offset],
                    &black_bishops_boards[kids_offset],
                    &black_knights_boards[kids_offset],
                    &black_rooks_boards[kids_offset],
                    &black_queens_boards[kids_offset],
                    &black_kings_boards[kids_offset],
                    current_player);
}

__global__ void run_first_stage_results(int * results,
                int level,
                short current_player,
                int * level_sizes,
                int * subtree_sizes,
                int * last,
                int basic_offset = 0) {
    int index = blockIdx.x * 1024 + threadIdx.x;
    if (index >= level_sizes[level]) return;
    int index_offset = (level == 0 ? 0 : subtree_sizes[level - 1]) + basic_offset;
    int kids_offset = subtree_sizes[level] + index * BOARDS_GENERATED + basic_offset;
    current_player ^= (level & 1);

    DBG(if (index != 0) {return;})
    // DBG(printf("Zbieram wyniki gracza %d od pozycji %d i current_player %d i czy maksymalizuje? %d\n", 
    //     index + index_offset, kids_offset, current_player, current_player == WHITE));

    gather_results(&results[index + index_offset], &results[kids_offset], current_player == WHITE, last);
}

__global__ void run_first_stage_evaluate(pos64 * white_pawns_boards,
                pos64 * white_bishops_boards,
                pos64 * white_knights_boards,
                pos64 * white_rooks_boards,
                pos64 * white_queens_boards,
                pos64 * white_kings_boards,
                pos64 * black_pawns_boards,
                pos64 * black_bishops_boards,
                pos64 * black_knights_boards,
                pos64 * black_rooks_boards,
                pos64 * black_queens_boards,
                pos64 * black_kings_boards,
                int * level_sizes,
                int * subtree_sizes,
                int * results,
                int basic_offset = 0) {
    int level = MAX_DEPTH - FIRST_STAGE_DEPTH;
    int index = blockIdx.x * 1024 + threadIdx.x;
    if (index >= level_sizes[level]) return;
    int index_offset = (level == 0 ? 0 : subtree_sizes[level - 1]) + basic_offset;
    
    DBG(if (index != 0) {return;})

    if ((white_kings_boards[index + index_offset] | black_kings_boards[index + index_offset]) == 0) {
        results[index + index_offset] = INF;    
    }
    else {
        results[index + index_offset] = evaluate_position(white_pawns_boards[index + index_offset],
                                                        white_bishops_boards[index + index_offset],
                                                        white_knights_boards[index + index_offset],
                                                        white_rooks_boards[index + index_offset],
                                                        white_queens_boards[index + index_offset],
                                                        white_kings_boards[index + index_offset],
                                                        black_pawns_boards[index + index_offset], 
                                                        black_bishops_boards[index + index_offset], 
                                                        black_knights_boards[index + index_offset],
                                                        black_rooks_boards[index + index_offset],
                                                        black_queens_boards[index + index_offset],
                                                        black_kings_boards[index + index_offset]);
    }
    DBG(printf("Evaluuje [%d] %d i mam %d\n", index, index + index_offset, results[index + index_offset]));
}

__global__ void copy_result(int * results, int from, int to) {
    DBG(printf("Kopiuje wynik z %d do %d\n", from, to));
    results[to] = results[from];
}

__global__ void copy_data(pos64 * white_pawns_boards,
                pos64 * white_bishops_boards,
                pos64 * white_knights_boards,
                pos64 * white_rooks_boards,
                pos64 * white_queens_boards,
                pos64 * white_kings_boards,
                pos64 * black_pawns_boards,
                pos64 * black_bishops_boards,
                pos64 * black_knights_boards,
                pos64 * black_rooks_boards,
                pos64 * black_queens_boards,
                pos64 * black_kings_boards,
                int from, int to) {
    DBG(printf("Kopiuje dane z %d do %d\n", from, to));
    white_pawns_boards[to] = white_pawns_boards[from];
    white_bishops_boards[to] = white_bishops_boards[from];
    white_knights_boards[to] = white_knights_boards[from];
    white_rooks_boards[to] = white_rooks_boards[from];
    white_queens_boards[to] = white_queens_boards[from];
    white_kings_boards[to] = white_kings_boards[from];
    black_pawns_boards[to] = black_pawns_boards[from];
    black_bishops_boards[to] = black_bishops_boards[from];
    black_knights_boards[to] = black_knights_boards[from];
    black_rooks_boards[to] = black_rooks_boards[from];
    black_queens_boards[to] = black_queens_boards[from];
    black_kings_boards[to] = black_kings_boards[from];
}

void search(const short& current_player,
            const int& move_num,
            pos64& white_pawns,
            pos64& white_bishops,
            pos64& white_knights,
            pos64& white_rooks,
            pos64& white_queens,
            pos64& white_kings,
            pos64& black_pawns,
            pos64& black_bishops,
            pos64& black_knights,
            pos64& black_rooks,
            pos64& black_queens,
            pos64& black_kings) {
        
    pos64*  white_pawns_boards;
    pos64*  white_bishops_boards;
    pos64*  white_knights_boards;
    pos64*  white_rooks_boards;
    pos64*  white_queens_boards;
    pos64*  white_kings_boards;
    pos64*  black_pawns_boards;
    pos64*  black_bishops_boards;
    pos64*  black_knights_boards;
    pos64*  black_rooks_boards;
    pos64*  black_queens_boards;
    pos64*  black_kings_boards;
    int * results;
    CHECK_ALLOC(hipMalloc(&white_pawns_boards, sizeof(pos64) * MAX_BOARDS_IN_MEMORY));
    CHECK_ALLOC(hipMalloc(&white_bishops_boards, sizeof(pos64) * MAX_BOARDS_IN_MEMORY));
    CHECK_ALLOC(hipMalloc(&white_knights_boards, sizeof(pos64) * MAX_BOARDS_IN_MEMORY));
    CHECK_ALLOC(hipMalloc(&white_rooks_boards, sizeof(pos64) * MAX_BOARDS_IN_MEMORY));
    CHECK_ALLOC(hipMalloc(&white_queens_boards, sizeof(pos64) * MAX_BOARDS_IN_MEMORY));
    CHECK_ALLOC(hipMalloc(&white_kings_boards, sizeof(pos64) * MAX_BOARDS_IN_MEMORY));
    CHECK_ALLOC(hipMalloc(&black_pawns_boards, sizeof(pos64) * MAX_BOARDS_IN_MEMORY));
    CHECK_ALLOC(hipMalloc(&black_bishops_boards, sizeof(pos64) * MAX_BOARDS_IN_MEMORY));
    CHECK_ALLOC(hipMalloc(&black_knights_boards, sizeof(pos64) * MAX_BOARDS_IN_MEMORY));
    CHECK_ALLOC(hipMalloc(&black_rooks_boards, sizeof(pos64) * MAX_BOARDS_IN_MEMORY));
    CHECK_ALLOC(hipMalloc(&black_queens_boards, sizeof(pos64) * MAX_BOARDS_IN_MEMORY));
    CHECK_ALLOC(hipMalloc(&black_kings_boards, sizeof(pos64) * MAX_BOARDS_IN_MEMORY));
    CHECK_ALLOC(hipMalloc(&results, sizeof(int) * MAX_BOARDS_IN_MEMORY));

    init_searching<<<1, 1>>>(white_pawns_boards,
                white_bishops_boards,
                white_knights_boards,
                white_rooks_boards,
                white_queens_boards,
                white_kings_boards,
                black_pawns_boards,
                black_bishops_boards,
                black_knights_boards,
                black_rooks_boards,
                black_queens_boards,
                black_kings_boards,
                white_pawns,
                white_bishops,
                white_knights,
                white_rooks,
                white_queens,
                white_kings,
                black_pawns,
                black_bishops,
                black_knights,
                black_rooks,
                black_queens,
                black_kings);
    gpuErrchk(hipDeviceSynchronize());
    gpuErrchk(hipPeekAtLastError());

    // generating moves in first stage
    DBG(printf("Stage 1 - generating moves\n"));
    for (int i = 0; i < FIRST_STAGE_DEPTH; i++) {
        run_first_stage<<<BLOCKS, THREADS>>>(white_pawns_boards,
                    white_bishops_boards,
                    white_knights_boards,
                    white_rooks_boards,
                    white_queens_boards,
                    white_kings_boards,
                    black_pawns_boards,
                    black_bishops_boards,
                    black_knights_boards,
                    black_rooks_boards,
                    black_queens_boards,
                    black_kings_boards,
                    i,
                    current_player,
                    d_level_sizes,
                    d_subtree_sizes);
        gpuErrchk(hipDeviceSynchronize());
        gpuErrchk(hipPeekAtLastError());
    }
    DBG(printf("Stage finished successfully\n"));

    // second stage
    int basic_offset = h_subtree_sizes[FIRST_STAGE_DEPTH] + 1;
    int player_offset = h_subtree_sizes[FIRST_STAGE_DEPTH - 1];
    for (int o = 0; o < h_level_sizes[FIRST_STAGE_DEPTH - 1]; o++) {

        copy_data<<<1, 1>>>(white_pawns_boards,
                    white_bishops_boards,
                    white_knights_boards,
                    white_rooks_boards,
                    white_queens_boards,
                    white_kings_boards,
                    black_pawns_boards,
                    black_bishops_boards,
                    black_knights_boards,
                    black_rooks_boards,
                    black_queens_boards,
                    black_kings_boards, 
                    player_offset + o,
                    basic_offset);
        gpuErrchk(hipDeviceSynchronize());
        gpuErrchk(hipPeekAtLastError());

        DBG(printf("Stage 2 - generating moves\n"));
        //generating moves
        for (int i = 0; i < MAX_DEPTH - FIRST_STAGE_DEPTH; i++) {
            run_first_stage<<<BLOCKS, THREADS>>>(white_pawns_boards,
                    white_bishops_boards,
                    white_knights_boards,
                    white_rooks_boards,
                    white_queens_boards,
                    white_kings_boards,
                    black_pawns_boards,
                    black_bishops_boards,
                    black_knights_boards,
                    black_rooks_boards,
                    black_queens_boards,
                    black_kings_boards,
                    i,
                    current_player ^ (i & 1) ^ ((FIRST_STAGE_DEPTH + i) & 1),
                    d_level_sizes,
                    d_subtree_sizes,
                    basic_offset);
            gpuErrchk(hipDeviceSynchronize());
            gpuErrchk(hipPeekAtLastError());
        }
        DBG(printf("Stage finished successfully\n"));

        DBG(printf("Stage 2 - evaluating\n"));
        // evaluating
        run_first_stage_evaluate<<<BLOCKS, THREADS>>>(white_pawns_boards,
                    white_bishops_boards,
                    white_knights_boards,
                    white_rooks_boards,
                    white_queens_boards,
                    white_kings_boards,
                    black_pawns_boards,
                    black_bishops_boards,
                    black_knights_boards,
                    black_rooks_boards,
                    black_queens_boards,
                    black_kings_boards,
                    d_level_sizes,
                    d_subtree_sizes,
                    results,
                    basic_offset);
        gpuErrchk(hipDeviceSynchronize());
        gpuErrchk(hipPeekAtLastError());
        DBG(printf("Stage finished successfully\n"));

        DBG(printf("Stage 2 - gathering results\n"));
        // gathering results
        for (int i = MAX_DEPTH - FIRST_STAGE_DEPTH - 1; i >= 0 ; i--) {
             run_first_stage_results<<<BLOCKS, THREADS>>>(results,
                    i,
                    current_player ^ (i & 1) ^ ((FIRST_STAGE_DEPTH + i) & 1),
                    d_level_sizes,
                    d_subtree_sizes,
                    last,
                    basic_offset);
            gpuErrchk(hipDeviceSynchronize());
            gpuErrchk(hipPeekAtLastError());   
        }
        DBG(printf("Stage finished successfully\n"));

        copy_result<<<1, 1>>>(results, basic_offset, player_offset + o);
        gpuErrchk(hipDeviceSynchronize());
        gpuErrchk(hipPeekAtLastError());
    }

    DBG(printf("Stage 1 - gathering results\n"));
    // acquiring results for first stage
     for (int i = FIRST_STAGE_DEPTH; i >= 0; i--) {
        run_first_stage_results<<<BLOCKS, THREADS>>>(results,
                    i,
                    current_player,
                    d_level_sizes,
                    d_subtree_sizes,
                    last);
        gpuErrchk(hipDeviceSynchronize());
        gpuErrchk(hipPeekAtLastError());
    }
    DBG(printf("Stage finished successfully\n"));
    
    end_searching(
        &white_pawns,
        &white_bishops,
        &white_knights,
        &white_rooks,
        &white_queens,
        &white_kings,
        &black_pawns,
        &black_bishops,
        &black_knights,
        &black_rooks,
        &black_queens,
        &black_kings,
        white_pawns_boards,
        white_bishops_boards,
        white_knights_boards,
        white_rooks_boards,
        white_queens_boards,
        white_kings_boards,
        black_pawns_boards,
        black_bishops_boards,
        black_knights_boards,
        black_rooks_boards,
        black_queens_boards,
        black_kings_boards,
        last);

    hipFree(white_pawns_boards);
    hipFree(white_bishops_boards);
    hipFree(white_knights_boards);
    hipFree(white_rooks_boards);
    hipFree(white_queens_boards);
    hipFree(white_kings_boards);
    hipFree(black_pawns_boards);
    hipFree(black_bishops_boards);
    hipFree(black_knights_boards);
    hipFree(black_rooks_boards);
    hipFree(black_queens_boards);
    hipFree(black_kings_boards);
    hipFree(results);
}




















