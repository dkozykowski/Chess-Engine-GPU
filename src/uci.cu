#include "hip/hip_runtime.h"
#include <cassert>
#include <iostream>
#include <sstream>
#include <string>

#include "evaluate.cuh"
#include "macros.cuh"
#include "moves.cuh"
#include "position.cuh"
#include "search.cuh"
#include "uci.cuh"

namespace UCI {

pos64 whitePawns;
pos64 whiteBishops;
pos64 whiteKnights;
pos64 whiteRooks;
pos64 whiteQueens;
pos64 whiteKings;
pos64 blackPawns;
pos64 blackBishops;
pos64 blackKnights;
pos64 blackRooks;
pos64 blackQueens;
pos64 blackKings;

void newgame(short &currentPlayer, int &moveNum);
void move(std::istringstream &is, short &currentPlayer, int &moveNum);
void printGame(short currentPlayer, int moveNum);
void printEval();
void go(short &currentPlayer, int &moveNum);
void printMoves(pos64 whitePawns, pos64 whiteBishops, pos64 whiteKnights,
                pos64 whiteRooks, pos64 whiteQueens, pos64 whiteKings,
                pos64 blackPawns, pos64 blackBishops, pos64 blackKnights,
                pos64 blackRooks, pos64 blackQueens, pos64 blackKings,
                short currentPlayer);
std::string getMoveString(pos64 currentPos, pos64 newPos);

int _log2(pos64 x);

void loop() {
    short currentPlayer;
    int moveNum;
    newgame(currentPlayer, moveNum);
    SEARCH::init();

    std::string token, cmd;

    do {
        if (!std::getline(std::cin, cmd)) break;

        std::istringstream is(cmd);

        token
            .clear();  // Avoid a stale if getline() returns empty or blank line
        is >> std::skipws >> token;

        if (token == "exit" || token == "quit" || token == "stop" ||
            token == "q")
            break;
        else if (token == "ucinewgame")
            newgame(currentPlayer, moveNum);
        else if (token == "d")
            printGame(currentPlayer, moveNum);
        else if (token == "flip")
            POSITION::flipPosition(whitePawns, whiteBishops, whiteKnights,
                                   whiteRooks, whiteQueens, whiteKings,
                                   blackPawns, blackBishops, blackKnights,
                                   blackRooks, blackQueens, blackKings);
        else if (token == "move")
            move(is, currentPlayer, moveNum);
        else if (token == "go")
            go(currentPlayer, moveNum);
        // else if (token == "bench")      bench(pos, is, states);
        else if (token == "eval")
            printEval();
        else if (token == "moves")
            printMoves(whitePawns, whiteBishops, whiteKnights, whiteRooks,
                       whiteQueens, whiteKings, blackPawns, blackBishops,
                       blackKnights, blackRooks, blackQueens, blackKings,
                       currentPlayer);
        else
            std::cout << "Unknown command: " << cmd << std::endl;
    } while (true);

    SEARCH::terminate();
}

void newgame(short &currentPlayer, int &moveNum) {
    whitePawns = WHITE_PAWN_STARTING_POS;
    whiteBishops = WHITE_BISHOP_STARTING_POS;
    whiteKnights = WHITE_KNIGHT_STARTING_POS;
    whiteRooks = WHITE_ROOK_STARTING_POS;
    whiteQueens = WHITE_QUEEN_STARTING_POS;
    whiteKings = WHITE_KING_STARTING_POS;

    blackPawns = BLACK_PAWN_STARTING_POS;
    blackBishops = BLACK_BISHOP_STARTING_POS;
    blackKnights = BLACK_KNIGHT_STARTING_POS;
    blackRooks = BLACK_ROOK_STARTING_POS;
    blackQueens = BLACK_QUEEN_STARTING_POS;
    blackKings = BLACK_KING_STARTING_POS;

    currentPlayer = WHITE;
    moveNum = 0;
}

void move(std::istringstream &is, short &currentPlayer, int &moveNum) {
    std::string moveToken;
    is >> std::skipws >> moveToken;

    // validate
    if (moveToken.length() != 4) {
        printf("Invalid move\n");
        return;
    }

    int fromCol = moveToken[0] >= 'a' ? moveToken[0] - 'a' : moveToken[0] - 'A';
    int fromRow = moveToken[1] - '1';
    int toCol = moveToken[2] >= 'a' ? moveToken[2] - 'a' : moveToken[2] - 'A';
    int toRow = moveToken[3] - '1';

    if (fromCol < 0 || fromRow < 0 || toCol < 0 || toRow < 0 || 8 <= fromCol ||
        8 <= fromRow || 8 <= toCol || 8 <= toRow) {
        printf("Invalid move\n");
        return;
    }

    POSITION::moveChess(fromCol, fromRow, toCol, toRow, currentPlayer,
                        whitePawns, whiteBishops, whiteKnights, whiteRooks,
                        whiteQueens, whiteKings, blackPawns, blackBishops,
                        blackKnights, blackRooks, blackQueens, blackKings);
    moveNum++;
    currentPlayer ^= 1;
}

void printGame(short currentPlayer, int moveNum) {
    printf("Move number %d\n", moveNum);
    printf("Current player - %s\n", currentPlayer == WHITE ? "White" : "Black");
    POSITION::printPosition(whitePawns, whiteBishops, whiteKnights, whiteRooks,
                            whiteQueens, whiteKings, blackPawns, blackBishops,
                            blackKnights, blackRooks, blackQueens, blackKings);
}

__global__ void eval(int *result, pos64 whitePawns, pos64 whiteBishops,
                     pos64 whiteKnights, pos64 whiteRooks, pos64 whiteQueens,
                     pos64 whiteKings, pos64 blackPawns, pos64 blackBishops,
                     pos64 blackKnights, pos64 blackRooks, pos64 blackQueens,
                     pos64 blackKings) {
    *result = EVALUATION::evaluatePosition(
        whitePawns, whiteBishops, whiteKnights, whiteRooks, whiteQueens,
        whiteKings, blackPawns, blackBishops, blackKnights, blackRooks,
        blackQueens, blackKings);
}

void printEval() {
    int *dResult, *hResult;
    hResult = new int;
    hipMalloc(&dResult, sizeof(int));
    eval<<<1, 1>>>(dResult, whitePawns, whiteBishops, whiteKnights, whiteRooks,
                   whiteQueens, whiteKings, blackPawns, blackBishops,
                   blackKnights, blackRooks, blackQueens, blackKings);
    hipMemcpy(hResult, dResult, sizeof(int), hipMemcpyDeviceToHost);
    printf("Current evaluation from white side: %d\n", *hResult);
    delete hResult;
    hipFree(dResult);
}

void go(short &currentPlayer, int &moveNum) {
    pos64 *position = new pos64[12];
    position[WHITE_PAWN_OFFSET] = whitePawns;
    position[WHITE_BISHOP_OFFSET] = whiteBishops;
    position[WHITE_KNIGHT_OFFSET] = whiteKnights;
    position[WHITE_ROOK_OFFSET] = whiteRooks;
    position[WHITE_QUEEN_OFFSET] = whiteQueens;
    position[WHITE_KING_OFFSET] = whiteKings;
    position[BLACK_PAWN_OFFSET] = blackPawns;
    position[BLACK_BISHOP_OFFSET] = blackBishops;
    position[BLACK_KNIGHT_OFFSET] = blackKnights;
    position[BLACK_ROOK_OFFSET] = blackRooks;
    position[BLACK_QUEEN_OFFSET] = blackQueens;
    position[BLACK_KING_OFFSET] = blackKings;

    SEARCH::findBestMove(currentPlayer, position);

    pos64 new_whitePawns = position[WHITE_PAWN_OFFSET];
    pos64 new_whiteBishops = position[WHITE_BISHOP_OFFSET];
    pos64 new_whiteKnights = position[WHITE_KNIGHT_OFFSET];
    pos64 new_whiteRooks = position[WHITE_ROOK_OFFSET];
    pos64 new_whiteQueens = position[WHITE_QUEEN_OFFSET];
    pos64 new_whiteKings = position[WHITE_KING_OFFSET];
    pos64 new_blackPawns = position[BLACK_PAWN_OFFSET];
    pos64 new_blackBishops = position[BLACK_BISHOP_OFFSET];
    pos64 new_blackKnights = position[BLACK_KNIGHT_OFFSET];
    pos64 new_blackRooks = position[BLACK_ROOK_OFFSET];
    pos64 new_blackQueens = position[BLACK_QUEEN_OFFSET];
    pos64 new_blackKings = position[BLACK_KING_OFFSET];

    DBG2(POSITION::printPosition(
        new_whitePawns, new_whiteBishops, new_whiteKnights, new_whiteRooks,
        new_whiteQueens, new_whiteKings, new_blackPawns, new_blackBishops,
        new_blackKnights, new_blackRooks, new_blackQueens, new_blackKings));

    if (currentPlayer == WHITE) {
        pos64 currentPos = whitePawns | whiteBishops | whiteKnights |
                           whiteRooks | whiteQueens | whiteKings;
        pos64 newPos = new_whitePawns | new_whiteBishops | new_whiteKnights |
                       new_whiteRooks | new_whiteQueens | new_whiteKings;
        std::cout << getMoveString(currentPos, newPos) << "\n";
    } else if (currentPlayer == BLACK) {
        pos64 currentPos = blackPawns | blackBishops | blackKnights |
                           blackRooks | blackQueens | blackKings;
        pos64 newPos = new_blackPawns | new_blackBishops | new_blackKnights |
                       new_blackRooks | new_blackQueens | new_blackKings;
        std::cout << getMoveString(currentPos, newPos) << "\n";
    }
}

std::string getMoveString(pos64 currentPos, pos64 newPos) {
    pos64 diff = currentPos ^ newPos;
    pos64 from = currentPos & diff;
    pos64 to = newPos & diff;
    int from_pos = _log2(from);
    int to_pos = _log2(to);
    std::string result = "____";
    result[0] = from_pos % 8 + 'a';
    result[1] = from_pos / 8 + '1';
    result[2] = to_pos % 8 + 'a';
    result[3] = to_pos / 8 + '1';
    return result;
}

int _log2(pos64 x) {  // asserting x is a power of two
    for (int i = 0; i < x; i++) {
        if ((x & (((pos64)1) << i)) != 0) {
            return i;
        }
    }
    return 0;
}

void printMoves(pos64 whitePawns, pos64 whiteBishops, pos64 whiteKnights,
                pos64 whiteRooks, pos64 whiteQueens, pos64 whiteKings,
                pos64 blackPawns, pos64 blackBishops, pos64 blackKnights,
                pos64 blackRooks, pos64 blackQueens, pos64 blackKings,
                short currentPlayer) {
    pos64 *position = new pos64[12];
    pos64 *generatedBoards = new pos64[255 * BOARD_SIZE];

    position[WHITE_PAWN_OFFSET] = whitePawns;
    position[WHITE_BISHOP_OFFSET] = whiteBishops;
    position[WHITE_KNIGHT_OFFSET] = whiteKnights;
    position[WHITE_ROOK_OFFSET] = whiteRooks;
    position[WHITE_QUEEN_OFFSET] = whiteQueens;
    position[WHITE_KING_OFFSET] = whiteKings;
    position[BLACK_PAWN_OFFSET] = blackPawns;
    position[BLACK_BISHOP_OFFSET] = blackBishops;
    position[BLACK_KNIGHT_OFFSET] = blackKnights;
    position[BLACK_ROOK_OFFSET] = blackRooks;
    position[BLACK_QUEEN_OFFSET] = blackQueens;
    position[BLACK_KING_OFFSET] = blackKings;

    MOVES::generateMoves(position, generatedBoards, currentPlayer == WHITE);
    std::string any;
    for (int x = 0; x < 255; x++) {
        if (((generatedBoards + (x * BOARD_SIZE))[BLACK_KING_OFFSET] |
             (generatedBoards + (x * BOARD_SIZE))[WHITE_KING_OFFSET]) == 0)
            break;
        POSITION::printPosition(
            (generatedBoards + (x * BOARD_SIZE))[WHITE_PAWN_OFFSET],
            (generatedBoards + (x * BOARD_SIZE))[WHITE_BISHOP_OFFSET],
            (generatedBoards + (x * BOARD_SIZE))[WHITE_KNIGHT_OFFSET],
            (generatedBoards + (x * BOARD_SIZE))[WHITE_ROOK_OFFSET],
            (generatedBoards + (x * BOARD_SIZE))[WHITE_QUEEN_OFFSET],
            (generatedBoards + (x * BOARD_SIZE))[WHITE_KING_OFFSET],
            (generatedBoards + (x * BOARD_SIZE))[BLACK_PAWN_OFFSET],
            (generatedBoards + (x * BOARD_SIZE))[BLACK_BISHOP_OFFSET],
            (generatedBoards + (x * BOARD_SIZE))[BLACK_KNIGHT_OFFSET],
            (generatedBoards + (x * BOARD_SIZE))[BLACK_ROOK_OFFSET],
            (generatedBoards + (x * BOARD_SIZE))[BLACK_QUEEN_OFFSET],
            (generatedBoards + (x * BOARD_SIZE))[BLACK_KING_OFFSET]);

        std::getline(std::cin, any);
        if (any == "q") break;
    }

    free(position);
    free(generatedBoards);
}
}  // namespace UCI